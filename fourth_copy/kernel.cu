#include "headerfile.h"
#include "getvariables.h"
#include "initialise_and_addnoise.h"
#include "cudafiles_1.h"
#include "cudafiles_2.h"
#include "graph_terms.h"
// #include "diffusion.cuh"


int main(){

//set the initial latice spacing. 
latice_spacing_z = 1.0; 
previous_latice_spacing_z = latice_spacing_z;

//collect variables
getvariables();

//allocate memory on the machine
phi_A = (double*) malloc(volume*sizeof(double));
phi_B = (double*) malloc(volume*sizeof(double));
phi_C = (double*) malloc(volume*sizeof(double));

delta_phi_A_delta_tau = (double*) malloc(volume*sizeof(double));
delta_phi_B_delta_tau = (double*) malloc(volume*sizeof(double));
delta_phi_C_delta_tau = (double*) malloc(volume*sizeof(double));


lattice_height = (double*) malloc(volume*sizeof(double));


//create grid				
initialise();

// gives percentage of a or b polymer at each grid point
addnoise();


//initialise_solvent();
//this function adds the solvent to the system
		
//start calculation of system
phi_diffusion_calc();

//confirm completion
cout << "  " << endl;
cout << "Finished. Press enter to close window" << endl;

// pause
cin.get();

//end
}

void phi_diffusion_calc()
{   


	hipSetDevice(0);

    // allocate memory on the device
	hipMalloc((void **) &device_phi_A, volume*sizeof(double));
	hipMalloc((void **) &device_phi_B, volume*sizeof(double));
	hipMalloc((void **) &device_mu1_A, volume*sizeof(double));
	hipMalloc((void **) &device_mu2_A, volume*sizeof(double));
	hipMalloc((void **) &device_mu1_B, volume*sizeof(double));
	hipMalloc((void **) &device_mu2_B, volume*sizeof(double));
	hipMalloc((void **) &device_mu1_A_grad_lambda_x, volume*sizeof(double));
	hipMalloc((void **) &device_mu2_A_grad_lambda_x, volume*sizeof(double));
	hipMalloc((void **) &device_mu1_B_grad_lambda_x, volume*sizeof(double));
	hipMalloc((void **) &device_mu2_B_grad_lambda_x, volume*sizeof(double));
	hipMalloc((void **) &device_mu1_A_grad_lambda_y, volume*sizeof(double));
	hipMalloc((void **) &device_mu2_A_grad_lambda_y, volume*sizeof(double));
	hipMalloc((void **) &device_mu1_B_grad_lambda_y, volume*sizeof(double));
	hipMalloc((void **) &device_mu2_B_grad_lambda_y, volume*sizeof(double));
	hipMalloc((void **) &device_mu1_A_grad_lambda_z, volume*sizeof(double));
	hipMalloc((void **) &device_mu2_A_grad_lambda_z, volume*sizeof(double));
	hipMalloc((void **) &device_mu1_B_grad_lambda_z, volume*sizeof(double));
	hipMalloc((void **) &device_mu2_B_grad_lambda_z, volume*sizeof(double));

	hipMalloc((void **) &device_delta_phi_A_delta_tau, volume*sizeof(double));
	hipMalloc((void **) &device_delta_phi_B_delta_tau, volume*sizeof(double));
	hipMalloc((void **) &device_deltas, volume*sizeof(double));
	hipMalloc((void **) &device_lattice_height, volume*sizeof(double));



    // copy phi matrice from the host to the device
    hipMemcpy(device_phi_A, phi_A, volume*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_phi_B, phi_B, volume*sizeof(double), hipMemcpyHostToDevice);


    // defines the number of grids and threads to be used in the parallel computation
    // both grids and threads can have 3 dimensions, but number of threads is limited to 512
    // so best to use two block dimensions of size length each and one thread dimension of size length
	dim3 phi_diffusion_grid(phi_diffusion_num_blocks_x, phi_diffusion_num_blocks_y, 1);
	dim3 phi_diffusion_threads(phi_diffusion_num_threads_x, 1, 1);   
	dim3 phi_surface_grid(phi_diffusion_num_blocks_x, 1, 1);
	dim3 phi_surface_threads(phi_diffusion_num_threads_x, 1, 1);
	dim3 two_d_threads(phi_diffusion_num_blocks_x, 1, 1);
	dim3 two_d_blocks(phi_diffusion_num_blocks_y, 1, 1);


	//load variables that will run time estimation can be made
	time_t start,end;
    double dif;
	double timesofar = 0;
	double runtime;
	double percentcomplete;
	 
	//apply solvent to entire system
	initialise_solvent <<< phi_diffusion_grid, phi_diffusion_threads, 0 >>>(device_phi_A, device_phi_B, solvent); 

	//give each point a delta_z value depending on its solvent concentration
	make_deltas <<< phi_diffusion_grid, phi_diffusion_threads, 0 >>>(delta_x, device_phi_A, device_phi_B, device_deltas, solvent);

	//find the height of each point using the delta_z values
	//WARNING!!! THIS BIT OF CODE IS INEFFICIENT
	find_z_height <<< phi_diffusion_grid, phi_diffusion_threads, 0 >>>(device_deltas, device_lattice_height);
	

	//copy array from GPU to CPU
	hipMemcpy(phi_A, device_phi_A, volume*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(phi_B, device_phi_B, volume*sizeof(double), hipMemcpyDeviceToHost);
	

	//copy lattice heights from GPU to CPU
	hipMemcpy(lattice_height, device_lattice_height, volume*sizeof(double), hipMemcpyDeviceToHost);
	
	//store the initital data to files and subfolders
	store_data();
	new_store_data();
	check_data();
	//store_minimum_data();


	//begin the main loop 

	for (counter = counter_start+1; counter < countmax; counter++)
	{  
		
		//record time so time estimation can be made
		time (&start);
		
		
		//begin inner loop that iterates the system between each data output
		for (int loop_counter = 0; loop_counter < num_saves; loop_counter++)
		{   


			//remove a small amount of solvent from the top layer					 
			remove_top_solvent <<< phi_diffusion_grid, phi_diffusion_threads, 0 >>>(device_phi_A, device_phi_B, evaporation);
		
			//give each point a delta_z value depending on its solvent concentration
	        make_deltas <<< phi_diffusion_grid, phi_diffusion_threads, 0 >>>(delta_x, device_phi_A, device_phi_B, device_deltas, solvent);

			//employ surface conditions on the system
			phi_surf <<< phi_surface_grid, phi_surface_threads, 0 >>>(device_phi_A, device_phi_B, device_deltas); 

			//calcualte the mu values		
			mu_calc <<< phi_diffusion_grid, phi_diffusion_threads, 0 >>>
				   (device_phi_A,
				    device_phi_B,
					device_mu1_A,
					device_mu2_A,
					device_mu1_B,
					device_mu2_B,
					delta_x,
					delta_y,
					kappa,
					chi_scale,
					pbc_x,
					pbc_y,
					pbc_z,
					deg_polyA,
					deg_polyB,
					deg_polyC,
					kappa,
					kai_AB,
					kai_AC,
					kai_BC,
					device_deltas
					);
	
		

			//employ surface conditions on the mu array to match the system
			mu_surf <<< phi_surface_grid, phi_surface_threads, 0 >>>
				    (device_phi_A,
					device_phi_B,
					device_mu1_A,
					device_mu2_A,
					device_mu1_B,
					device_mu2_B,
					g_upper,
					h_upper,
					g_lower,
					h_lower
					);
	    
	 	 
			//begin the diffusion calculation
			new_phi_diffusion_1 <<< phi_diffusion_grid, phi_diffusion_threads, 0 >>>
				                   (device_phi_A,
								    device_phi_B,
									device_mu1_A,
									device_mu2_A,
									device_mu1_B,
									device_mu2_B,
									device_mu1_A_grad_lambda_x,
									device_mu2_A_grad_lambda_x,
									device_mu1_B_grad_lambda_x,
									device_mu2_B_grad_lambda_x,
									device_mu1_A_grad_lambda_y,
									device_mu2_A_grad_lambda_y,
									device_mu1_B_grad_lambda_y,
									device_mu2_B_grad_lambda_y,
									device_mu1_A_grad_lambda_z,
									device_mu2_A_grad_lambda_z,
									device_mu1_B_grad_lambda_z,
									device_mu2_B_grad_lambda_z,
									delta_x,
									delta_y,
									delta_t,
									mobility,
									pbc_x,
									pbc_y,
									pbc_z,
									kappa,
									device_deltas
									);


		    //finish the diffusion calculation
			new_phi_diffusion_2 <<< phi_diffusion_grid, phi_diffusion_threads, 0 >>>
		
		                           (device_phi_A,
								    device_phi_B,
									device_mu1_A_grad_lambda_x,
									device_mu2_A_grad_lambda_x,
									device_mu1_B_grad_lambda_x,
									device_mu2_B_grad_lambda_x,
									device_mu1_A_grad_lambda_y,
									device_mu2_A_grad_lambda_y,
									device_mu1_B_grad_lambda_y,
									device_mu2_B_grad_lambda_y,
									device_mu1_A_grad_lambda_z,
									device_mu2_A_grad_lambda_z,
									device_mu1_B_grad_lambda_z,
									device_mu2_B_grad_lambda_z,
									delta_x,
									delta_y,
									delta_t,
									mobility,
									pbc_x,
									pbc_y,
									pbc_z,
									kappa,
									device_delta_phi_A_delta_tau,
									device_delta_phi_B_delta_tau,
									device_deltas
						            );
		    
			//find the height of each lattice point 
			find_z_height <<< phi_diffusion_grid, phi_diffusion_threads, 0 >>>(device_deltas, device_lattice_height);
		    
			//put the new array into the interpolated array
			
		}


		// copy the phi matrix values back from the device to the host
		hipMemcpy(phi_A, device_phi_A, volume*sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(phi_B, device_phi_B, volume*sizeof(double), hipMemcpyDeviceToHost);

		
		// copy lattice height values back from the device to the host
		hipMemcpy(lattice_height, device_lattice_height, volume*sizeof(double), hipMemcpyDeviceToHost);

		
		//store the initital data to files and subfolders 
	    store_data();
		new_store_data();
		check_data();
		//store_minimum_data();

		
		
		/*
		//this code outputs the can between each output values and is useful for debugging.
		hipMemcpy(delta_phi_A_delta_tau, device_delta_phi_A_delta_tau, volume*sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(delta_phi_B_delta_tau, device_delta_phi_B_delta_tau, volume*sizeof(double), hipMemcpyDeviceToHost);
		store_deltas();
		*/
						
		
		
		//work out time estimation
		time (&end);
        dif = difftime (end,start) / 60;
		runtime = dif*(countmax-1);
		timesofar += dif;
		percentcomplete = 100 * timesofar/runtime;

		
		//print time estimation to screen
		cout << "loop " << counter << " of " << countmax-1 << " took " << dif <<" minutes." << endl;
		cout << "times run so far is " << timesofar << " minutes of a estimated time of " << runtime << " minutes." << endl;
		cout << percentcomplete << "% complete..." << endl;


     //end the outer loop
   	}

   	
	//free the memory on the GPU
	hipFree(device_phi_A);
	hipFree(device_phi_B);
	hipFree(device_mu1_A);
	hipFree(device_mu2_A);
	hipFree(device_mu1_B);
	hipFree(device_mu2_B);
	
	
	hipFree(device_mu1_A_grad_lambda_x);
	hipFree(device_mu2_A_grad_lambda_x);
	hipFree(device_mu1_B_grad_lambda_x);
	hipFree(device_mu2_B_grad_lambda_x);
	hipFree(device_mu1_A_grad_lambda_y);
	hipFree(device_mu2_A_grad_lambda_y);
	hipFree(device_mu1_B_grad_lambda_y);
	hipFree(device_mu2_B_grad_lambda_y);
	hipFree(device_mu1_A_grad_lambda_z);
	hipFree(device_mu2_A_grad_lambda_z);
	hipFree(device_mu1_B_grad_lambda_z);
	hipFree(device_mu2_B_grad_lambda_z);
	hipFree(device_deltas);


	//free the allocted memory on the host
	free(phi_A);
	free(phi_B);
	free(phi_C);

	free(array_delta_z);
	free(lattice_height);
	
	//thats the end
}





