#include "headerfile.h"
#include "getvariables.h"
#include "initialise_and_addnoise.h"
#include "cudafiles_1.h"
#include "cudafiles_2.h"
#include "graph_terms.h"
// #include "diffusion.cuh"


int main(){

//set the initial latice spacing. 
latice_spacing_z = 1.0; 
previous_latice_spacing_z = latice_spacing_z;

//collect variables
getvariables();

//allocate memory on the machine
phi_A = (double*) malloc(volume*sizeof(double));
phi_B = (double*) malloc(volume*sizeof(double));
phi_C = (double*) malloc(volume*sizeof(double));

delta_phi_A_delta_tau = (double*) malloc(volume*sizeof(double));
delta_phi_B_delta_tau = (double*) malloc(volume*sizeof(double));
delta_phi_C_delta_tau = (double*) malloc(volume*sizeof(double));



phi_z = (double*) malloc(length_z*sizeof(double));
	    

//create grid				
initialise();

// gives percentage of a or b polymer at each grid point
addnoise();


//initialise_solvent();
//this function adds the solvent to the system
		
//start calculation of system
phi_diffusion_calc();

//confirm completion
cout << "  " << endl;
cout << "Finished. Press enter to close window" << endl;

// pause
cin.get();

//end
}

void phi_diffusion_calc()
{   


	hipSetDevice(0);

    // allocate memory on the device
	hipMalloc((void **) &device_phi_A, volume*sizeof(double));
	hipMalloc((void **) &device_phi_B, volume*sizeof(double));
	hipMalloc((void **) &device_mu1_A, volume*sizeof(double));
	hipMalloc((void **) &device_mu2_A, volume*sizeof(double));
	hipMalloc((void **) &device_mu1_B, volume*sizeof(double));
	hipMalloc((void **) &device_mu2_B, volume*sizeof(double));
	hipMalloc((void **) &device_mu1_A_grad_lambda_x, volume*sizeof(double));
	hipMalloc((void **) &device_mu2_A_grad_lambda_x, volume*sizeof(double));
	hipMalloc((void **) &device_mu1_B_grad_lambda_x, volume*sizeof(double));
	hipMalloc((void **) &device_mu2_B_grad_lambda_x, volume*sizeof(double));
	hipMalloc((void **) &device_mu1_A_grad_lambda_y, volume*sizeof(double));
	hipMalloc((void **) &device_mu2_A_grad_lambda_y, volume*sizeof(double));
	hipMalloc((void **) &device_mu1_B_grad_lambda_y, volume*sizeof(double));
	hipMalloc((void **) &device_mu2_B_grad_lambda_y, volume*sizeof(double));
	hipMalloc((void **) &device_mu1_A_grad_lambda_z, volume*sizeof(double));
	hipMalloc((void **) &device_mu2_A_grad_lambda_z, volume*sizeof(double));
	hipMalloc((void **) &device_mu1_B_grad_lambda_z, volume*sizeof(double));
	hipMalloc((void **) &device_mu2_B_grad_lambda_z, volume*sizeof(double));

	hipMalloc((void **) &device_delta_phi_A_delta_tau, volume*sizeof(double));
	hipMalloc((void **) &device_delta_phi_B_delta_tau, volume*sizeof(double));

	hipMalloc((void **) &device_phisum2d_A, length_y*length_z*sizeof(double));
	hipMalloc((void **) &device_phisum2d_B, length_y*length_z*sizeof(double));
	hipMalloc((void **) &device_phisum1d_A, length_z*sizeof(double));
	hipMalloc((void **) &device_phisum1d_B, length_z*sizeof(double));


	hipMalloc((void **) &device_delta_z, length_z*sizeof(double));


    // copy phi matrice from the host to the device
    hipMemcpy(device_phi_A, phi_A, volume*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_phi_B, phi_B, volume*sizeof(double), hipMemcpyHostToDevice);

    // defines the number of grids and threads to be used in the parallel computation
    // both grids and threads can have 3 dimensions, but number of threads is limited to 512
    // so best to use two block dimensions of size length each and one thread dimension of size length

	
	dim3 phi_diffusion_grid(phi_diffusion_num_blocks_x, phi_diffusion_num_blocks_y, 1);
	dim3 phi_diffusion_threads(phi_diffusion_num_threads_x, 1, 1);   
	dim3 phi_surface_grid(phi_diffusion_num_blocks_x, 1, 1);
	dim3 phi_surface_threads(phi_diffusion_num_threads_x, 1, 1);
	
	dim3 two_d_threads(phi_diffusion_num_blocks_x, 1, 1);
	dim3 two_d_blocks(phi_diffusion_num_blocks_y, 1, 1);


	time_t start,end;
    double dif;
	double timesofar = 0;
	double runtime;
	double percentcomplete;
	 

	initialise_solvent <<< phi_diffusion_grid, phi_diffusion_threads, 0 >>>(device_phi_A, device_phi_B, solvent); 

	phi_sum2d <<< phi_diffusion_grid, phi_diffusion_threads, 0 >>>(device_phi_A, device_phi_B, device_phisum2d_A, device_phisum2d_B);
	phi_sum1d <<< two_d_blocks, two_d_threads, 0 >>>(device_phisum1d_A, device_phisum1d_B, device_phisum2d_A, device_phisum2d_B, device_delta_z);


	hipMemcpy(phi_A, device_phi_A, volume*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(phi_B, device_phi_B, volume*sizeof(double), hipMemcpyDeviceToHost);
	
	//store_minimum_data();
	store_data();
	check_data();

	
	array_delta_z = (double*) malloc(length_z*sizeof(double));

	hipMemcpy(array_delta_z, device_delta_z, length_z*sizeof(double), hipMemcpyDeviceToHost);
	print_values_z();
	
	


	for (counter = counter_start+1; counter < countmax; counter++)
	{  
		
		time (&start);
		
			   
		for (int loop_counter = 0; loop_counter < num_saves; loop_counter++)
		{   


			// kenrel invocation
       
			//phi_surf <<< phi_surface_grid, phi_surface_threads, 0 >>>(device_phi_A, device_phi_B, device_delta_z); 
	   
					 
			remove_top_solvent <<< phi_diffusion_grid, phi_diffusion_threads, 0 >>>(device_phi_A, device_phi_B, evaporation);
		
		
			phi_sum2d <<< phi_diffusion_grid, phi_diffusion_threads, 0 >>>(device_phi_A, device_phi_B, device_phisum2d_A, device_phisum2d_B);
		
		
			phi_sum1d <<< two_d_blocks, two_d_threads, 0 >>>(device_phisum1d_A, device_phisum1d_B, device_phisum2d_A, device_phisum2d_B, device_delta_z);


			phi_surf <<< phi_surface_grid, phi_surface_threads, 0 >>>(device_phi_A, device_phi_B, device_delta_z); 

			//kernel invocation
		
			mu_calc <<< phi_diffusion_grid, phi_diffusion_threads, 0 >>>
				   (device_phi_A,
				    device_phi_B,
					device_mu1_A,
					device_mu2_A,
					device_mu1_B,
					device_mu2_B,
					delta_x,
					delta_y,
				//	delta_z,
					kappa,
					chi_scale,
					pbc_x,
					pbc_y,
					pbc_z,
					deg_polyA,
					deg_polyB,
					deg_polyC,
					kappa,
					kai_AB,
					kai_AC,
					kai_BC,
					device_delta_z);
	
		

       
			mu_surf <<< phi_surface_grid, phi_surface_threads, 0 >>>
				    (device_phi_A,
					device_phi_B,
					device_mu1_A,
					device_mu2_A,
					device_mu1_B,
					device_mu2_B,
					g_upper,
					h_upper,
					g_lower,
					h_lower);
	    


	 	 
			new_phi_diffusion_1 <<< phi_diffusion_grid, phi_diffusion_threads, 0 >>>
				                   (device_phi_A,
								    device_phi_B,
									device_mu1_A,
									device_mu2_A,
									device_mu1_B,
									device_mu2_B,
									device_mu1_A_grad_lambda_x,
									device_mu2_A_grad_lambda_x,
									device_mu1_B_grad_lambda_x,
									device_mu2_B_grad_lambda_x,
									device_mu1_A_grad_lambda_y,
									device_mu2_A_grad_lambda_y,
									device_mu1_B_grad_lambda_y,
									device_mu2_B_grad_lambda_y,
									device_mu1_A_grad_lambda_z,
									device_mu2_A_grad_lambda_z,
									device_mu1_B_grad_lambda_z,
									device_mu2_B_grad_lambda_z,
									delta_x,
									delta_y,
								//	delta_z,
									delta_t,
									mobility,
									pbc_x,
									pbc_y,
									pbc_z,
									kappa,
									device_delta_z
									);


		
			new_phi_diffusion_2 <<< phi_diffusion_grid, phi_diffusion_threads, 0 >>>
		
		                           (device_phi_A,
								    device_phi_B,
									device_mu1_A_grad_lambda_x,
									device_mu2_A_grad_lambda_x,
									device_mu1_B_grad_lambda_x,
									device_mu2_B_grad_lambda_x,
									device_mu1_A_grad_lambda_y,
									device_mu2_A_grad_lambda_y,
									device_mu1_B_grad_lambda_y,
									device_mu2_B_grad_lambda_y,
									device_mu1_A_grad_lambda_z,
									device_mu2_A_grad_lambda_z,
									device_mu1_B_grad_lambda_z,
									device_mu2_B_grad_lambda_z,
									delta_x,
									delta_y,
								//	delta_z,
									delta_t,
									mobility,
									pbc_x,
									pbc_y,
									pbc_z,
									kappa,
									device_delta_phi_A_delta_tau,
									device_delta_phi_B_delta_tau,
									device_delta_z
						            );
		
		}


		// copy the phi matrix values back from the device to the host
		hipMemcpy(phi_A, device_phi_A, volume*sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(phi_B, device_phi_B, volume*sizeof(double), hipMemcpyDeviceToHost);

		

		//printf("stopped");
		//cin.get();

	    // graphplot_text();
		// graphplot_visual();
		// areaplot_text();

	    store_data();
		
		check_data();
		//store_minimum_data();

		
		/*

		hipMemcpy(delta_phi_A_delta_tau, device_delta_phi_A_delta_tau, volume*sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(delta_phi_B_delta_tau, device_delta_phi_B_delta_tau, volume*sizeof(double), hipMemcpyDeviceToHost);
		//store_deltas();

		hipMemcpy(array_delta_z, device_delta_z, length_z*sizeof(double), hipMemcpyDeviceToHost);
	    print_values_z();
				
		*/
		

		time (&end);
        dif = difftime (end,start) / 60;
		runtime = dif*(countmax-1);
		timesofar += dif;
		percentcomplete = 100 * timesofar/runtime;

		

		cout << "loop " << counter << " of " << countmax-1 << " took " << dif <<" minutes." << endl;
		cout << "times run so far is " << timesofar << " minutes of a estimated time of " << runtime << " minutes." << endl;
		cout << percentcomplete << "% complete..." << endl;


   	}
   	
	hipFree(device_phi_A);
	hipFree(device_phi_B);
	hipFree(device_mu1_A);
	hipFree(device_mu2_A);
	hipFree(device_mu1_B);
	hipFree(device_mu2_B);
	
	hipFree(device_mu1_A_grad_lambda_x);
	hipFree(device_mu2_A_grad_lambda_x);
	hipFree(device_mu1_B_grad_lambda_x);
	hipFree(device_mu2_B_grad_lambda_x);
	hipFree(device_mu1_A_grad_lambda_y);
	hipFree(device_mu2_A_grad_lambda_y);
	hipFree(device_mu1_B_grad_lambda_y);
	hipFree(device_mu2_B_grad_lambda_y);
	hipFree(device_mu1_A_grad_lambda_z);
	hipFree(device_mu2_A_grad_lambda_z);
	hipFree(device_mu1_B_grad_lambda_z);
	hipFree(device_mu2_B_grad_lambda_z);

	hipFree(device_phisum2d_A);
	hipFree(device_phisum2d_B);
	hipFree(device_phisum1d_A);
	hipFree(device_phisum1d_B);


	hipFree(device_delta_z);

	free(phi_A);
	free(phi_B);
	free(phi_C);
	free(phi_z);


	free(array_delta_z);
}





